/**
 * @brief
 * ragged
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <hipcub/hipcub.hpp>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/ragged_utils.h"

namespace {


}  // namespace k2
