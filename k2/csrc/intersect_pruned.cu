#include "hip/hip_runtime.h"
/**
 * @brief
 * compose
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <limits>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/hash.h"
#include "k2/csrc/ragged_ops.h"

namespace k2 {

namespace intersect_pruned_internal {

/* Information associated with a state active on a particular frame..  */
struct StateInfo {
  /* abs_state_id is the state-index in a_fsas_.  Note: the ind0 in here
     won't necessarily match the ind0 within FrameInfo::state if
     a_fsas_stride_ == 0. */
  int32_t a_fsas_state_idx01;

  /* Caution: this is ACTUALLY A FLOAT that has been bit-twiddled using
     FloatToOrderedInt/OrderedIntToFloat so we can use atomic max.  It
     represents a Viterbi-style 'forward probability'.  (Viterbi, meaning: we
     use max not log-sum).  You can take the pruned lattice and rescore it if
     you want log-sum.  */
  int32_t forward_loglike;

  /* Note: this `backward_loglike` is the best score of any path from here to
     the end, minus the best path in the overall FSA, i.e. it's the backward
     score you get if, at the final-state, you set backward_loglike ==
     forward_loglike. So backward_loglike + OrderedIntToFloat(forward_loglike)
     <= 0, and you can treat it somewhat like a posterior (except they don't sum
     to one as we're using max, not log-add).
  */
  float backward_loglike;
};

struct ArcInfo {              // for an arc that wasn't pruned away...
  int32_t a_fsas_arc_idx012;  // the arc-index in a_fsas_.
  float arc_loglike;          // loglike on this arc: equals loglike from data
  // (nnet output, == b_fsas), plus loglike from
  // the arc in a_fsas.

  union {
    // these 3 different ways of storing the index of the destination state
    // are used at different stages of the algorithm; we give them different
    // names for clarity.
    int32_t dest_a_fsas_state_idx01;  // The destination-state as an index
    // into a_fsas_.
    int32_t dest_info_state_idx1;  // The destination-state as an idx1 into the
                                   // next FrameInfo's `arcs` or `states`,
                                   // omitting the FSA-index which can be worked
                                   // out from the structure of this frame's
                                   // ArcInfo.
  } u;
  float end_loglike;  // loglike at the end of the arc just before
  // (conceptually) it joins the destination state.
};

static std::ostream &operator<<(std::ostream &os, const StateInfo &s) {
  os << "StateInfo{" << s.a_fsas_state_idx01 << ","
     << OrderedIntToFloat(s.forward_loglike) << "," << s.backward_loglike
     << "}";
  return os;
}

static std::ostream &operator<<(std::ostream &os, const ArcInfo &a) {
  os << "ArcInfo{" << a.a_fsas_arc_idx012 << "," << a.arc_loglike << ","
     << a.u.dest_a_fsas_state_idx01 << "," << a.end_loglike << "}";
  return os;
}


}  // namespace intersect_pruned_internal

using namespace intersect_pruned_internal;  // NOLINT

// Caution: this is really a .cu file.  It contains mixed host and device code.

/*
   Pruned intersection (a.k.a. composition) that corresponds to decoding for
   speech recognition-type tasks.  Can use either different decoding graphs (one
   per acoustic sequence) or a shared graph
*/
class MultiGraphDenseIntersectPruned {
 public:
  /**
     Pruned intersection (a.k.a. composition) that corresponds to decoding for
     speech recognition-type tasks

       @param [in] a_fsas  The decoding graphs, one per sequence.  E.g. might
                           just be a linear sequence of phones, or might be
                           something more complicated.  Must have either the
                           same Dim0() as b_fsas, or Size0()==1 in which
                           case the graph is shared.
       @param [in] b_fsas  The neural-net output, with each frame containing the
                           log-likes of each phone.  A series of sequences of
                           (in general) different length.
       @param [in] search_beam    "Default" search/decoding beam.  The actual
                           beam is dynamic and also depends on max_active and
                           min_active.
       @param [in] output_beam    Beam for pruning the output FSA, will
                                  typically be smaller than search_beam.
       @param [in] min_active  Minimum number of FSA states that are allowed to
                           be active on any given frame for any given
                           intersection/composition task. This is advisory,
                           in that it will try not to have fewer than this
                           number active.
       @param [in] max_active  Maximum number of FSA states that are allowed to
                           be active on any given frame for any given
                           intersection/composition task. This is advisory,
                           in that it will try not to exceed that but may not
                           always succeed.  This determines the hash size.
   */
  MultiGraphDenseIntersectPruned(FsaVec &a_fsas, DenseFsaVec &b_fsas,
                                 float search_beam, float output_beam,
                                 int32_t min_active, int32_t max_active)
      : a_fsas_(a_fsas),
        b_fsas_(b_fsas),
        search_beam_(search_beam),
        output_beam_(output_beam),
        min_active_(min_active),
        max_active_(max_active),
        dynamic_beams_(a_fsas.Context(), b_fsas.shape.Dim0(), search_beam) {
    NVTX_RANGE(K2_FUNC);
    c_ = GetContext(a_fsas.shape, b_fsas.shape);
    K2_CHECK(b_fsas.scores.IsContiguous());
    K2_CHECK_GT(search_beam, 0);
    K2_CHECK_GT(output_beam, 0);
    K2_CHECK_GE(min_active, 0);
    K2_CHECK_GT(max_active, min_active);
    K2_CHECK(a_fsas.shape.Dim0() == b_fsas.shape.Dim0() ||
             a_fsas.shape.Dim0() == 1);
    K2_CHECK_GE(b_fsas.shape.Dim0(), 1);
    int32_t num_seqs = b_fsas.shape.Dim0();

    int32_t num_buckets = RoundUpToNearestPowerOfTwo(num_seqs * 4 *
                                                     max_active);
    if (num_buckets < 128)
      num_buckets = 128;
    state_map_ = Hash32(c_, num_buckets);
    int32_t num_a_copies;
    if (a_fsas.shape.Dim0() == 1) {
      a_fsas_stride_ = 0;
      state_map_fsa_stride_ = a_fsas.TotSize(1);
      num_a_copies = b_fsas.shape.Dim0();
    } else {
      K2_CHECK_EQ(a_fsas.shape.Dim0(), b_fsas.shape.Dim0());
      a_fsas_stride_ = 1;
      state_map_fsa_stride_ = 0;
      num_a_copies = 1;
    }
    int64_t num_keys = num_a_copies * (int64_t)a_fsas.TotSize(1);
    K2_CHECK(num_keys == (uint32_t)num_keys);
  }

  // The information we have for each frame of the pruned-intersection (really:
  // decoding) algorithm.  We keep an array of these, one for each frame, up to
  // the length of the longest sequence we're decoding plus one.
  struct FrameInfo {
    // States that are active at the beginning of this frame.  Indexed
    // [fsa_idx][state_idx], where fsa_idx indexes b_fsas_ (and a_fsas_, if
    // a_fsas_stride_ != 0); and state_idx just enumerates the active states
    // on this frame (as state_idx01's in a_fsas_).
    Ragged<StateInfo> states;  // 2 axes: fsa, state

    // Indexed [fsa_idx][state_idx][arc_idx].. the first 2 indexes are
    // the same as those into 'states' (the first 2 levels of the structure
    // are shared), and the last one enumerates the arcs leaving each of those
    // states.
    //
    // Note: there may be indexes [fsa_idx] that have no states (because that
    // FSA had fewer frames than the max), and indexes [fsa_idx][state_idx] that
    // have no arcs due to pruning.
    Ragged<ArcInfo> arcs;  // 3 axes: fsa, state, arc
  };

  /* Does the main work of intersection/composition, but doesn't produce any
     output; the output is provided when you call FormatOutput(). */
  void Intersect() {
    /*
      T is the largest number of (frames+1) of neural net output, or the largest
      number of frames of log-likelihoods we count the final frame with (0,
      -inf, -inf..) that is used for the final-arcc.  The largest number of
      states in the fsas represented by b_fsas equals T+1 (e.g. 1 frame would
      require 2 states, because that 1 frame is the arc from state 0 to state
      1).  So the #states is 2 greater than the actual number of frames in the
      neural-net output.
    */
    NVTX_RANGE(K2_FUNC);
    T_ = b_fsas_.shape.MaxSize(1);
    int32_t num_fsas = b_fsas_.shape.Dim0(), T = T_;

    std::ostringstream os;
    os << "Intersect:T=" << T << ",num_fsas=" << num_fsas
       << ",TotSize(1)=" << b_fsas_.shape.TotSize(1);
    NVTX_RANGE(os.str().c_str());

    // we'll initially populate frames_[0.. T+1], but discard the one at T+1,
    // which has no arcs or states, the ones we use are from 0 to T.
    frames_.reserve(T + 2);

    frames_.push_back(InitialFrameInfo());

    for (int32_t t = 0; t <= T; t++) {
      frames_.push_back(PropagateForward(t, frames_.back().get()));
    }
    // The FrameInfo for time T+1 will have no states.  We did that
    // last PropagateForward so that the 'arcs' member of frames_[T]
    // is set up (it has no arcs but we need the shape).
    frames_.pop_back();

    SetBackwardProbsFinal(frames_[T].get());
    for (int32_t t = T - 1; t >= 0; t--) {
      PropagateBackwardAndPrune(t, frames_[t].get(),
                                frames_[t + 1].get());
    }
  }

  // Return FrameInfo for 1st frame, with `states` set but `arcs` not set.
  std::unique_ptr<FrameInfo> InitialFrameInfo() {
    NVTX_RANGE("InitialFrameInfo");
    int32_t num_fsas = b_fsas_.shape.Dim0();
    std::unique_ptr<FrameInfo> ans = std::make_unique<FrameInfo>();

    if (a_fsas_.Dim0() == 1) {
      int32_t start_states_per_seq = (a_fsas_.shape.TotSize(1) > 0),  // 0 or 1
          num_start_states = num_fsas * start_states_per_seq;
      ans->states = Ragged<StateInfo>(
          RegularRaggedShape(c_, num_fsas, start_states_per_seq),
          Array1<StateInfo>(c_, num_start_states));
      StateInfo *states_data = ans->states.values.Data();
      K2_EVAL(
          c_, num_start_states, lambda_set_states, (int32_t i)->void {
            StateInfo info;
            info.a_fsas_state_idx01 = 0;  // start state of a_fsas_
            info.forward_loglike = FloatToOrderedInt(0.0);
            states_data[i] = info;
          });
    } else {
      Ragged<int32_t> start_states = GetStartStates(a_fsas_);
      ans->states =
          Ragged<StateInfo>(start_states.shape,
                            Array1<StateInfo>(c_, start_states.NumElements()));
      StateInfo *ans_states_values_data = ans->states.values.Data();
      const int32_t *start_states_values_data = start_states.values.Data(),
                    *start_states_row_ids1_data =
                        start_states.shape.RowIds(1).Data();
      K2_EVAL(
          c_, start_states.NumElements(), lambda_set_state_info,
          (int32_t states_idx01)->void {
            StateInfo info;
            info.a_fsas_state_idx01 = start_states_values_data[states_idx01];
            info.forward_loglike = FloatToOrderedInt(0.0);
            ans_states_values_data[states_idx01] = info;
          });
    }
    return ans;
  }

  void FormatOutput(FsaVec *ofsa, Array1<int32_t> *arc_map_a,
                    Array1<int32_t> *arc_map_b) {
    NVTX_RANGE("FormatOutput");

    int32_t T = T_;


    ContextPtr c_cpu = GetCpuContext();
    Array1<ArcInfo *> arcs_data_ptrs(c_cpu, T + 1);
    Array1<int32_t *> arcs_row_splits1_ptrs(c_cpu, T + 1);
    for (int32_t t = 0; t <= T; t++) {
      arcs_data_ptrs.Data()[t] = frames_[t]->arcs.values.Data();
      arcs_row_splits1_ptrs.Data()[t] = frames_[t]->arcs.RowSplits(1).Data();
    }
    // transfer to GPU if we're using a GPU
    arcs_data_ptrs = arcs_data_ptrs.To(c_);
    ArcInfo **arcs_data_ptrs_data = arcs_data_ptrs.Data();
    arcs_row_splits1_ptrs = arcs_row_splits1_ptrs.To(c_);
    int32_t **arcs_row_splits1_ptrs_data = arcs_row_splits1_ptrs.Data();
    const int32_t *b_fsas_row_splits1 = b_fsas_.shape.RowSplits(1).Data();
    const int32_t *a_fsas_row_splits1 = a_fsas_.RowSplits(1).Data();
    int32_t a_fsas_stride = a_fsas_stride_;  // 0 or 1 depending if the decoding
                                             // graph is shared.
    int32_t num_fsas = b_fsas_.shape.Dim0();

    RaggedShape final_arcs_shape;
    { /*  This block populates `final_arcs_shape`.  It is the shape of a ragged
          tensor of arcs that conceptually would live at frames_[T+1]->arcs.  It
          contains no actual arcs, but may contain some states, that represent
          "missing" final-states.  The problem we are trying to solve is that
          there was a start-state for an FSA but no final-state because it did
          not survive pruning, and this could lead to an output FSA that is
          invalid or is misinterpreted (because we are interpreting a non-final
          state as a final state).
       */
      Array1<int32_t> num_extra_states(c_, num_fsas + 1);
      int32_t *num_extra_states_data = num_extra_states.Data();
      K2_EVAL(c_, num_fsas, lambda_set_num_extra_states, (int32_t i) -> void {
          int32_t final_t = b_fsas_row_splits1[i+1] - b_fsas_row_splits1[i];
          int32_t *arcs_row_splits1_data = arcs_row_splits1_ptrs_data[final_t];
          int32_t num_states_final_t = arcs_row_splits1_data[i + 1] -
                                       arcs_row_splits1_data[i];
          K2_CHECK_LE(num_states_final_t, 1);

          // has_start_state is 1 if there is a start-state; note, we don't prune
          // the start-states, so they'll be present if they were present in a_fsas_.
          int32_t has_start_state = (a_fsas_row_splits1[i * a_fsas_stride] <
                                     a_fsas_row_splits1[i * a_fsas_stride + 1]);

          // num_extra_states_data[i] will be 1 if there was a start state but no final-state;
          // else, 0.
          num_extra_states_data[i] = has_start_state * (1 - num_states_final_t);
        });
      K2_LOG(INFO) << "num_extra_states = " << num_extra_states;
      ExclusiveSum(num_extra_states, &num_extra_states);

      RaggedShape top_shape = RaggedShape2(&num_extra_states, nullptr, -1),
               bottom_shape = RegularRaggedShape(c_, top_shape.NumElements(), 0);
      final_arcs_shape = ComposeRaggedShapes(top_shape, bottom_shape);
    }


    RaggedShape oshape;
    // see documentation of Stack() in ragged_ops.h for explanation.
    Array1<uint32_t> oshape_merge_map;

    {
      NVTX_RANGE("InitOshape");
      // each of these have 3 axes.
      std::vector<RaggedShape *> arcs_shapes(T + 2);
      for (int32_t t = 0; t <= T; t++)
        arcs_shapes[t] = &(frames_[t]->arcs.shape);
      arcs_shapes[T + 1] = &final_arcs_shape;

      // oshape is a 4-axis ragged tensor which is indexed:
      //   oshape[fsa_index][t][state_idx][arc_idx]
      int32_t axis = 1;
      oshape = Stack(axis, T + 2, arcs_shapes.data(), &oshape_merge_map);
    }


    int32_t *oshape_row_ids3 = oshape.RowIds(3).Data(),
            *oshape_row_ids2 = oshape.RowIds(2).Data(),
            *oshape_row_ids1 = oshape.RowIds(1).Data(),
            *oshape_row_splits3 = oshape.RowSplits(3).Data(),
            *oshape_row_splits2 = oshape.RowSplits(2).Data(),
            *oshape_row_splits1 = oshape.RowSplits(1).Data();


    int32_t num_arcs = oshape.NumElements();
    *arc_map_a = Array1<int32_t>(c_, num_arcs);
    *arc_map_b = Array1<int32_t>(c_, num_arcs);
    int32_t *arc_map_a_data = arc_map_a->Data(),
            *arc_map_b_data = arc_map_b->Data();
    Array1<Arc> arcs_out(c_, num_arcs);
    Arc *arcs_out_data = arcs_out.Data();
    const Arc *a_fsas_arcs = a_fsas_.values.Data();
    int32_t b_fsas_num_cols = b_fsas_.scores.Dim1();
    const int32_t *b_fsas_row_ids1 = b_fsas_.shape.RowIds(1).Data();

    const uint32_t *oshape_merge_map_data = oshape_merge_map.Data();

    K2_EVAL(
        c_, num_arcs, lambda_format_arc_data,
        (int32_t oarc_idx0123)->void {  // by 'oarc' we mean arc with shape `oshape`.
          int32_t oarc_idx012 = oshape_row_ids3[oarc_idx0123],
                   oarc_idx01 = oshape_row_ids2[oarc_idx012],
                    oarc_idx0 = oshape_row_ids1[oarc_idx01],
                   oarc_idx0x = oshape_row_splits1[oarc_idx0],
                  oarc_idx0xx = oshape_row_splits2[oarc_idx0x],
                    oarc_idx1 = oarc_idx01 - oarc_idx0x,
             oarc_idx01x_next = oshape_row_splits2[oarc_idx01 + 1];

          int32_t m = oshape_merge_map_data[oarc_idx0123],
                  t = m % (T + 2),  // actually we won't get t == T or t == T + 1
                                    // here since those frames have no arcs.
        arcs_idx012 = m / (T + 2);  // arc_idx012 into FrameInfo::arcs on time t,
                                    // index of the arc on that frame.

          K2_CHECK_EQ(t, oarc_idx1);

          const ArcInfo *arcs_data = arcs_data_ptrs_data[t];

          ArcInfo arc_info = arcs_data[arcs_idx012];
          Arc arc;
          arc.src_state = oarc_idx012 - oarc_idx0xx;
          // Note: the idx1 w.r.t. the frame's `arcs` is an idx2 w.r.t. `oshape`.
          int32_t dest_state_idx012 = oarc_idx01x_next +
                                      arc_info.u.dest_info_state_idx1;
          arc.dest_state = dest_state_idx012 - oarc_idx0xx;
          arc.label = a_fsas_arcs[arc_info.a_fsas_arc_idx012].label;

          int32_t fsa_id = oarc_idx0,
            b_fsas_idx0x = b_fsas_row_splits1[fsa_id],
            b_fsas_idx01 = b_fsas_idx0x + t,
             b_fsas_idx2 = (arc.label + 1),
       b_fsas_arc_idx012 = b_fsas_idx01 * b_fsas_num_cols + b_fsas_idx2;

          arc.score = arc_info.arc_loglike;
          arc_map_a_data[oarc_idx0123] = arc_info.a_fsas_arc_idx012;
          arc_map_b_data[oarc_idx0123] = b_fsas_arc_idx012;
          arcs_out_data[oarc_idx0123] = arc;
        });

    // Remove axis 1, which corresponds to time.
    *ofsa = FsaVec(RemoveAxis(oshape, 1), arcs_out);
  }

  /*
    Computes pruning cutoffs for this frame: these are the cutoffs for the arc
    "forward score", one per FSA.  This is a dynamic process involving
    dynamic_beams_ which are updated on each frame (they start off at
    search_beam_).

       @param [in] arc_end_scores  The "forward log-probs" (scores) at the
                    end of each arc, i.e. its contribution to the following
                    state.  Is a tensor indexed [fsa_id][state][arc]; we
                    will get rid of the [state] dim, combining it with the
                    [arc] dim, so it's just [fsa_id][arc]
                    It is conceptually unchanged by this operation but non-const
                    because row-ids of its shape may need to be generated.
       @return      Returns a vector of log-likelihood cutoffs, one per FSA (the
                    cutoff will be -infinity for FSAs that don't have any active
                    states).  The cutoffs will be of the form: the best score
                    for any arc, minus the dynamic beam.  See the code for how
                    the dynamic beam is adjusted; it will approach
                    'search_beam_' as long as the number of active states in
                    each FSA is between min_active and max_active.
  */
  Array1<float> GetPruningCutoffs(Ragged<float> &arc_end_scores) {
    NVTX_RANGE(K2_FUNC);
    int32_t num_fsas = arc_end_scores.shape.Dim0();

    // get the maximum score from each sub-list (i.e. each FSA, on this frame).
    // Note: can probably do this with a cub Reduce operation using an operator
    // that has side effects (that notices when it's operating across a
    // boundary).
    // the max will be -infinity for any FSA-id that doesn't have any active
    // states (e.g. because that stream has finished).
    // Casting to ragged2 just considers the top 2 indexes, ignoring the 3rd.
    // i.e. it's indexed by [fsa_id][state].
    Ragged<float> end_scores_per_fsa = arc_end_scores.RemoveAxis(1);
    Array1<float> max_per_fsa(c_, end_scores_per_fsa.Dim0());
    MaxPerSublist(end_scores_per_fsa, -std::numeric_limits<float>::infinity(),
                  &max_per_fsa);
    const int32_t *arc_end_scores_row_splits1_data =
        arc_end_scores.RowSplits(1).Data();
    const float *max_per_fsa_data = max_per_fsa.Data();
    float *dynamic_beams_data = dynamic_beams_.Data();

    float default_beam = search_beam_, max_active = max_active_,
          min_active = min_active_;
    K2_CHECK_LT(min_active, max_active);

    Array1<float> cutoffs(c_, num_fsas);
    float *cutoffs_data = cutoffs.Data();

    K2_EVAL(
        c_, num_fsas, lambda_set_beam_and_cutoffs, (int32_t i)->void {
          float best_loglike = max_per_fsa_data[i],
                dynamic_beam = dynamic_beams_data[i];
          int32_t active_states = arc_end_scores_row_splits1_data[i + 1] -
                                  arc_end_scores_row_splits1_data[i];
          if (active_states <= max_active) {
            // Not constrained by max_active...
            if (active_states >= min_active || active_states == 0) {
              // Neither the max_active nor min_active constraints
              // apply.  Gradually approach 'beam'
              // (Also approach 'beam' if active_states == 0; we might as
              // well, since there is nothing to prune here).
              dynamic_beam = 0.8 * dynamic_beam + 0.2 * default_beam;
            } else {
              // We violated the min_active constraint -> increase beam
              if (dynamic_beam < default_beam) dynamic_beam = default_beam;
              // gradually make the beam larger as long
              // as we are below min_active
              dynamic_beam *= 1.25;
            }
          } else {
            // We violated the max_active constraint -> decrease beam
            if (dynamic_beam > default_beam) dynamic_beam = default_beam;
            // Decrease the beam as long as we have more than
            // max_active active states.
            dynamic_beam *= 0.8;
          }
          dynamic_beams_data[i] = dynamic_beam;
          cutoffs_data[i] = best_loglike - dynamic_beam;
        });

    return cutoffs;
  }

  /*
    Returns list of arcs on this frame, consisting of all arcs leaving
    the states active on 'cur_frame'.

       @param [in] t       The time-index (on which to look up log-likes),
                           t >= 0
       @param [in] cur_frame   The FrameInfo for the current frame; only its
                       'states' member is expected to be set up on entry.
   */
  Ragged<ArcInfo> GetArcs(int32_t t, FrameInfo *cur_frame) {
    NVTX_RANGE(K2_FUNC);
    Ragged<StateInfo> &states = cur_frame->states;
    const StateInfo *state_values = states.values.Data();

    // in a_fsas_ (the decoding graphs), maps from state_idx01 to arc_idx01x.
    const int32_t *fsa_arc_splits = a_fsas_.shape.RowSplits(2).Data();

    int32_t num_states = states.values.Dim();
    Array1<int32_t> num_arcs(c_, num_states + 1);
    int32_t *num_arcs_data = num_arcs.Data();
    // `num_arcs` gives the num-arcs for each state in `states`.
    K2_EVAL(
        c_, num_states, num_arcs_lambda, (int32_t state_idx01)->void {
          int32_t a_fsas_state_idx01 =
                      state_values[state_idx01].a_fsas_state_idx01,
                  a_fsas_arc_idx01x = fsa_arc_splits[a_fsas_state_idx01],
                  a_fsas_arc_idx01x_next =
                      fsa_arc_splits[a_fsas_state_idx01 + 1],
                  a_fsas_num_arcs = a_fsas_arc_idx01x_next - a_fsas_arc_idx01x;
          num_arcs_data[state_idx01] = a_fsas_num_arcs;
        });
    ExclusiveSum(num_arcs, &num_arcs);

    // initialize shape of array that will hold arcs leaving the active states.
    // Its shape is [fsa_index][state][arc]; the top two levels are shared with
    // `states`.  'ai' means ArcInfo.
    RaggedShape ai_shape =
        ComposeRaggedShapes(states.shape, RaggedShape2(&num_arcs, nullptr, -1));

    // from state_idx01 (into `states` or `ai_shape`) -> fsa_idx0
    const int32_t *ai_row_ids1 = ai_shape.RowIds(1).Data();
    // from arc_idx012 (into `ai_shape`) to state_idx01
    const int32_t *ai_row_ids2 = ai_shape.RowIds(2).Data();
    // from state_idx01 to arc_idx01x
    const int32_t *ai_row_splits2 = ai_shape.RowSplits(2).Data();
    // from state_idx01 (into a_fsas_) to arc_idx01x (into a_fsas_)
    const int32_t *a_fsas_row_splits2 = a_fsas_.shape.RowSplits(2).Data();

    const Arc *arcs = a_fsas_.values.Data();
    // fsa_idx0 to ind0x (into b_fsas_), which gives the 1st row for this
    // sequence.
    const int32_t *b_fsas_row_ids1 = b_fsas_.shape.RowIds(1).Data();
    const int32_t *b_fsas_row_splits1 = b_fsas_.shape.RowSplits(1).Data();
    const float *score_data = b_fsas_.scores.Data();
    int32_t scores_num_cols = b_fsas_.scores.Dim1();
    auto scores_acc = b_fsas_.scores.Accessor();

    Ragged<ArcInfo> ai(ai_shape);
    ArcInfo *ai_data = ai.values.Data();  // uninitialized

    K2_EVAL(
        c_, ai.values.Dim(), ai_lambda, (int32_t ai_arc_idx012)->void {
          int32_t ai_state_idx01 = ai_row_ids2[ai_arc_idx012],
                  ai_fsa_idx0 = ai_row_ids1[ai_state_idx01],
                  ai_arc_idx01x = ai_row_splits2[ai_state_idx01],
                  ai_arc_idx2 = ai_arc_idx012 - ai_arc_idx01x;
          StateInfo sinfo = state_values[ai_state_idx01];
          int32_t a_fsas_arc_idx01x =
                      a_fsas_row_splits2[sinfo.a_fsas_state_idx01],
                  a_fsas_arc_idx012 = a_fsas_arc_idx01x + ai_arc_idx2;
          Arc arc = arcs[a_fsas_arc_idx012];

          int32_t scores_idx0x = b_fsas_row_splits1[ai_fsa_idx0],
                  scores_idx01 = scores_idx0x + t,  // t == ind1 into 'scores'
              scores_idx2 =
                  arc.label + 1;  // the +1 is so that -1 can be handled
          K2_DCHECK_LT(static_cast<uint32_t>(scores_idx2),
                       static_cast<uint32_t>(scores_num_cols));
          float acoustic_score = scores_acc(scores_idx01, scores_idx2);
          ArcInfo ai;
          ai.a_fsas_arc_idx012 = a_fsas_arc_idx012;
          ai.arc_loglike = acoustic_score + arc.score;
          ai.end_loglike =
              OrderedIntToFloat(sinfo.forward_loglike) + ai.arc_loglike;
          // at least currently, the ArcInfo object's src_state and dest_state
          // are idx1's not idx01's, i.e. they don't contain the FSA-index,
          // where as the ai element is an idx01, so we need to do this to
          // convert to an idx01; this relies on the fact that
          // sinfo.abs_state_id == arc.src_state
          // + a_fsas_fsa_idx0x.
          ai.u.dest_a_fsas_state_idx01 =
              sinfo.a_fsas_state_idx01 + arc.dest_state - arc.src_state;
          ai_data[ai_arc_idx012] = ai;
        });
    return ai;
  }

  // Later we may choose to support b_fsas_.Dim0() == 1 and a_fsas_.Dim0() > 1,
  // and we'll have to change various bits of code for that to work.
  inline int32_t NumFsas() { return b_fsas_.shape.Dim0(); }

  /*
    Does the forward-propagation (basically: the decoding step) and
    returns a newly allocated FrameInfo* object for the next frame.

      @param [in] t   Time-step that we are processing arcs leaving from;
                   will be called with t=0, t=1, ...
      @param [in] cur_frame  FrameInfo object for the states corresponding to
                   time t; will have its 'states' member set up but not its
                   'arcs' member (this function will create that).
     @return  Returns FrameInfo object corresponding to time t+1; will have its
             'states' member set up but not its 'arcs' member.
   */
  std::unique_ptr<FrameInfo> PropagateForward(int32_t t, FrameInfo *cur_frame) {
    NVTX_RANGE("PropagateForward");
    int32_t num_fsas = NumFsas();
    // Ragged<StateInfo> &states = cur_frame->states;
    // arc_info has 3 axes: fsa_id, state, arc.
    cur_frame->arcs = GetArcs(t, cur_frame);
    Ragged<ArcInfo> &arc_info = cur_frame->arcs;

    ArcInfo *ai_data = arc_info.values.Data();
    Array1<float> ai_data_array1(c_, cur_frame->arcs.values.Dim());
    float *ai_data_array1_data = ai_data_array1.Data();
    K2_EVAL(
        c_, ai_data_array1.Dim(), lambda_set_ai_data,
        (int32_t i)->void { ai_data_array1_data[i] = ai_data[i].end_loglike; });
    Ragged<float> ai_loglikes(arc_info.shape, ai_data_array1);

    // `cutoffs` is of dimension num_fsas.
    Array1<float> cutoffs = GetPruningCutoffs(ai_loglikes);
    if (t % 100 == 0)
      K2_LOG(INFO) << "Dynamic_beams = " << dynamic_beams_;
    float *cutoffs_data = cutoffs.Data();

    // write certain indexes ( into ai.values) to state_map_.Data().  Keeps
    // track of the active states and will allow us to assign a numbering to
    // them.
    int32_t *ai_row_ids1 = arc_info.shape.RowIds(1).Data(),
            *ai_row_ids2 = arc_info.shape.RowIds(2).Data();
    auto state_map_acc = state_map_.GetAccessor();
    int32_t state_map_fsa_stride = state_map_fsa_stride_;

    // renumber_states will be a renumbering that dictates which of the arcs in
    // 'ai' correspond to unique states.  Only one arc for each dest-state is
    // kept (it doesn't matter which one).
    Renumbering renumber_states(c_, arc_info.NumElements());
    char *keep_this_state_data = renumber_states.Keep().Data();


    {
      NVTX_RANGE("LambdaSetStateMap");
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_set_state_map,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]];
            int32_t dest_state_idx01 =
                ai_data[arc_idx012].u.dest_a_fsas_state_idx01;
            float end_loglike = ai_data[arc_idx012].end_loglike,
                  cutoff = cutoffs_data[fsa_id];
            char keep_this_state = 0;  // only one arc entering any state will
                                       // have its 'keep_this_state_data' entry
                                       // set to 1.
            if (end_loglike > cutoff) {
              int32_t state_map_idx = dest_state_idx01 +
                                      fsa_id * state_map_fsa_stride;
              if (state_map_acc.Insert(state_map_idx, arc_idx012))
                keep_this_state = 1;
            }
            keep_this_state_data[arc_idx012] = keep_this_state;
          });
    }


    int32_t num_states = renumber_states.NumNewElems();
    // state_reorder_data maps from (state_idx01 on next frame) to (the
    // arc_idx012 on this frame which is the source arc which we arbitrarily
    // choose as being "responsible" for the creation of that state).
    int32_t *state_reorder_data = renumber_states.Old2New().Data();

    // state_to_fsa_id maps from an index into the next frame's
    // FrameInfo::states.values() vector to the sequence-id (fsa_id) associated
    // with it.  It should be non-decreasing.
    Array1<int32_t> state_to_fsa_id(c_, num_states);
    {  // This block sets 'state_to_fsa_id'.
      NVTX_RANGE("LambdaSetStateToFsaId");
      int32_t *state_to_fsa_id_data = state_to_fsa_id.Data();
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_state_to_fsa_id,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]],
                    this_state_j = state_reorder_data[arc_idx012],
                    next_state_j = state_reorder_data[arc_idx012 + 1];
            if (next_state_j > this_state_j) {
              state_to_fsa_id_data[this_state_j] = fsa_id;
            }
          });

      K2_DCHECK(IsMonotonic(state_to_fsa_id));
    }

    std::unique_ptr<FrameInfo> ans = std::make_unique<FrameInfo>();
    Array1<int32_t> states_row_splits1(c_, num_fsas + 1);
    RowIdsToRowSplits(state_to_fsa_id, &states_row_splits1);
    ans->states = Ragged<StateInfo>(
        RaggedShape2(&states_row_splits1, &state_to_fsa_id, num_states),
        Array1<StateInfo>(c_, num_states));
    StateInfo *ans_states_data = ans->states.values.Data();
    const int32_t minus_inf_int =
        FloatToOrderedInt(-std::numeric_limits<float>::infinity());
    K2_EVAL(
        c_, num_states, lambda_init_loglike, (int32_t i)->void {
          ans_states_data[i].forward_loglike = minus_inf_int;
        });

    {
      NVTX_RANGE("LambdaModifyStateMap");
      // Modify the elements of `state_map` to refer to the indexes into
      // `ans->states` / `kept_states_data`, rather than the indexes into
      // ai_data. This will decrease some of the values in `state_map`, in
      // general.
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_modify_state_map,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]];
            int32_t dest_state_idx01 =
                ai_data[arc_idx012].u.dest_a_fsas_state_idx01;
            int32_t this_j = state_reorder_data[arc_idx012],
                    next_j = state_reorder_data[arc_idx012 + 1];
            if (next_j > this_j) {
              int32_t state_map_idx = dest_state_idx01 +
                                      fsa_id * state_map_fsa_stride;
              int32_t value, *value_addr;
              bool ans = state_map_acc.Find(state_map_idx,
                                            &value, &value_addr);
              K2_CHECK(ans);
              K2_CHECK_EQ(value, arc_idx012);
              // Note: this_j is an idx01 into ans->states.  previously it
              // contained an arc_idx012 (of the entering arc that won the
              // race).
              *value_addr = this_j;
            }
          });
    }

    // We'll set up the data of the kept states below...
    StateInfo *kept_states_data = ans->states.values.Data();

    {
      int32_t *ans_states_row_splits1_data = ans->states.RowSplits(1).Data();

      NVTX_RANGE("LambdaSetStates");
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_set_arcs_and_states,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]];

            ArcInfo &info = ai_data[arc_idx012];

            int32_t dest_a_fsas_state_idx01 = info.u.dest_a_fsas_state_idx01;


            int32_t state_map_idx = dest_a_fsas_state_idx01 +
                                    fsa_id * state_map_fsa_stride;
            int32_t state_idx01;
            if (!state_map_acc.Find(state_map_idx, &state_idx01))
              state_idx01 = -1;   // The destination state did not survive
                                  // pruning.

            int32_t state_idx1;
            if (state_idx01 >= 0) {
              int32_t state_idx0x = ans_states_row_splits1_data[fsa_id];
              state_idx1 = state_idx01 - state_idx0x;
            } else {
              state_idx1 = -1;  // Meaning: invalid.
            }
            // state_idx1 is the idx1 into ans->states, of the destination
            // state.
            info.u.dest_info_state_idx1 = state_idx1;
            if (state_idx1 < 0)
              return;

            // multiple threads may write the same value to the address written
            // to in the next line.
            kept_states_data[state_idx01].a_fsas_state_idx01 =
                dest_a_fsas_state_idx01;
            int32_t end_loglike_int = FloatToOrderedInt(info.end_loglike);
            // Set the forward log-like of the dest state to the largest of any
            // of those of the incoming arcs.  Note: we initialized this in
            // lambda_init_loglike above.
            atomicMax(&(kept_states_data[state_idx01].forward_loglike),
                      end_loglike_int);
          });
    }
    {
      NVTX_RANGE("LambdaResetStateMap");
      const int32_t *next_states_row_ids1 = ans->states.shape.RowIds(1).Data();
      K2_EVAL(
          c_, ans->states.NumElements(), lambda_reset_state_map,
          (int32_t state_idx01)->void {
            int32_t a_fsas_state_idx01 =
                        kept_states_data[state_idx01].a_fsas_state_idx01,
                fsa_idx0 = next_states_row_ids1[state_idx01];
            int32_t state_map_idx = a_fsas_state_idx01 +
                                    fsa_idx0 * state_map_fsa_stride;
            state_map_acc.Delete(state_map_idx);
          });
    }
    return ans;
  }


  /*
    Sets backward_loglike fields of StateInfo to the negative of the forward
    prob if (this is the final-state or !only_final_probs), else -infinity.

    This is used in computing the backward loglikes/scores for purposes of
    pruning.  This may be done after we're finished decoding/intersecting,
    or while we are still decoding.

    Note: something similar to this (setting backward-prob == forward-prob) is
    also done in PropagateBackward() when we detect final-states.  That's needed
    because not all sequences have the same length, so some may have reached
    their final state earlier.  (Note: we only get to the final-state of a_fsas_
    if we've reached the final frame of the input, because for non-final frames
    we always have -infinity as the log-prob corresponding to the symbol -1.)

    While we are still decoding, a background process will do pruning
    concurrently with the forward computation, for purposes of reducing memory
    usage (and so that most of the pruning can be made concurrent with the
    forward computation).  In this case we want to avoid pruning away anything
    that wouldn't have been pruned away if we were to have waited to the end;
    and it turns out that setting the backward probs to the negative of the
    forward probs (i.e.  for all states, not just final states) accomplishes
    this.  The issue was mentioned in the "Exact Lattice Generation.." paper and
    also in the code for Kaldi's lattice-faster-decoder; search for "As in [3],
    to save memory..."

      @param [in] cur_frame    Frame on which to set the backward probs
  */
  void SetBackwardProbsFinal(FrameInfo *cur_frame) {
    NVTX_RANGE("SetBackwardProbsFinal");
    Ragged<StateInfo> &cur_states = cur_frame->states;  // 2 axes: fsa,state
    int32_t num_states = cur_states.values.Dim();
    if (num_states == 0)
      return;
    StateInfo *cur_states_data = cur_states.values.Data();
    const int32_t *a_fsas_row_ids1_data = a_fsas_.shape.RowIds(1).Data(),
               *a_fsas_row_splits1_data = a_fsas_.shape.RowSplits(1).Data(),
              *cur_states_row_ids1_data = cur_states.RowIds(1).Data();
    double minus_inf = -std::numeric_limits<double>::infinity();

    K2_EVAL(c_, num_states, lambda_set_backward_prob, (int32_t state_idx01) -> void {
        StateInfo *info = cur_states_data + state_idx01;
        double backward_loglike,
            forward_loglike = OrderedIntToFloat(info->forward_loglike);
        if (forward_loglike - forward_loglike == 0) { // not -infinity...
          // canonically we'd set this to zero, but setting it to the forward
          // loglike when this is the final-state (in a_fsas_) has the effect of
          // making the (forward+backward) probs equivalent to the logprob minus
          // the best-path log-prob, which is convenient for pruning.  If this
          // is not actually the last frame of this sequence, which can happen
          // if this was called before the forward decoding process was
          // finished, what we are doing is a form of pruning that is guaranteed
          // not to prune anything out that would not have been pruned out if we
          // had waited until the real end of the file to do the pruning.
          backward_loglike = -forward_loglike;
        } else {
          backward_loglike = minus_inf;
        }
        info->backward_loglike = backward_loglike;
      });
    K2_LOG(INFO) << "states = " << cur_frame->states;
  }

  /*
    Does backward propagation of log-likes, which means setting the
    backward_loglike field of the StateInfo variable (for cur_frame); and
    prune/renumber the states of next_frame and the arcs of cur_frame.

    These backward log-likes are normalized in such a way that you can add them
    with the forward log-likes to produce the log-likelihood ratio vs the best
    path (this will be non-positive).  (To do this, for the final state we have
    to set the backward log-like to the negative of the forward log-like; see
    SetBackwardProbsFinal()).

    This function also prunes arc-indexes on `cur_frame` and state-indexes
    on `next_frame`.

       @param [in] t       The time-index (on which to look up log-likes),
                           t >= 0
       @param [in]  cur_frame The FrameInfo for the frame on which we want to
                              set the forward log-like, and prune the arcs.

       @param [in]  next_frame The next frame's FrameInfo; we will prune the
                             states on this frame (which also affects the
                             shape of the 'arcs').
                             Arcs on `cur_frame` have destination-states
                             on `next_frame`. The `backward_loglike` values
                             of states on `next_frame` are assumed to
                             already be set.
  */
  void PropagateBackwardAndPrune(int32_t t,
                                 FrameInfo *cur_frame,
                                 FrameInfo *next_frame) {
    NVTX_RANGE("PropagateBackwardAndPrune");
    int32_t num_states = cur_frame->states.NumElements(),
            num_arcs = cur_frame->arcs.NumElements();

    int32_t *a_fsas_row_ids1_data = a_fsas_.shape.RowIds(1).Data(),
            *a_fsas_row_splits1_data = a_fsas_.shape.RowSplits(1).Data();

    float minus_inf = -std::numeric_limits<float>::infinity();

    Ragged<float> arc_backward_prob(cur_frame->arcs.shape,
                                    Array1<float>(c_, cur_frame->arcs.NumElements()));
    float *arc_backward_prob_data = arc_backward_prob.values.Data();

    ArcInfo *ai_data = cur_frame->arcs.values.Data();
    int32_t *arcs_rowids1 = cur_frame->arcs.shape.RowIds(1).Data(),
            *arcs_rowids2 = cur_frame->arcs.shape.RowIds(2).Data(),
            *arcs_row_splits1 = cur_frame->arcs.shape.RowSplits(1).Data(),
            *arcs_row_splits2 = cur_frame->arcs.shape.RowSplits(2).Data();
    float output_beam = output_beam_;

    // compute arc backward probs, and set elements of 'keep_arcs'
    int32_t next_num_states = next_frame->states.TotSize(1);

    Renumbering renumber_cur_arcs(c_, num_arcs),
        renumber_next_states(c_, next_num_states);
    char *keep_cur_arcs_data = renumber_cur_arcs.Keep().Data(),
      *keep_next_states_data = renumber_next_states.Keep().Data();

    StateInfo *next_states_data = next_frame->states.values.Data();
    int32_t *next_arcs_row_splits2_data = next_frame->arcs.RowSplits(2).Data();
    K2_EVAL(c_, next_num_states, lambda_set_keep_next_states, (int32_t next_state_idx01) -> void {
        // Keep a state if there are arcs leaving it on the next frame (this is
        // required because we don't want to renumber the next frame's arcs); or
        // if its backward_loglike is nevertheless not minus_inf because it is a
        // final-state.
        keep_next_states_data[next_state_idx01] =
            ((next_arcs_row_splits2_data[next_state_idx01+1] !=
              next_arcs_row_splits2_data[next_state_idx01]) ||
             (next_states_data[next_state_idx01].backward_loglike != minus_inf));
      });

    K2_LOG(INFO) << "For t = " << t << ", renumber_next_states.Keep() = "
                 << renumber_next_states.Keep()
                 << ", next_states = " << next_frame->states;

    Array1<int32_t> next_states_row_splits1 = next_frame->states.RowSplits(1);
    const int32_t *next_states_row_splits1_data_old =
        next_states_row_splits1.Data();

    // Renumber the states on the next frame (note: the shape of the `states` is
    // the same as the 1st layer of the shape of the `arcs`).
    next_frame->arcs.shape = RemoveSomeEmptyLists(next_frame->arcs.shape, 1,
                                                  renumber_next_states);
    next_frame->states.shape = GetLayer(next_frame->arcs.shape, 0);
    next_frame->states.values = next_frame->states.values[renumber_next_states.New2Old()];

    StateInfo *next_states_data_new = next_frame->states.values.Data();

    const int32_t *next_states_row_splits1_data_new
        = next_frame->states.RowSplits(1).Data();

    int32_t *next_states_old2new_data = renumber_next_states.Old2New().Data();

    StateInfo *cur_states_data = cur_frame->states.values.Data();

    // next_states_row_splits1 maps from fsa_idx0 to state_idx01.
    int32_t *next_states_row_splits1_new =
        next_frame->states.shape.RowSplits(1).Data();

    //K2_EVAL(
    //    c_, num_arcs,
    //        lambda_set_arc_backward_prob_and_keep, (int32_t arcs_idx012)->void {

    auto lambda_set_arc_backward_prob_and_keep = [=]  __host__ __device__ (int32_t arcs_idx012) -> void {
          ArcInfo *arc = ai_data + arcs_idx012;
          int32_t state_idx01 = arcs_rowids2[arcs_idx012],
                     fsa_idx0 = arcs_rowids1[state_idx01],
          next_states_idx0x_old = next_states_row_splits1_data_old[fsa_idx0],
          next_states_idx0x_new = next_states_row_splits1_data_new[fsa_idx0];

          // `old` and `new` here are before and after pruning the state-ids of
          // the next frame, see `renumber_next_states`.
          // Note: if dest_state_idx1_old == -1, dest_state_idx01_old has a meaningless
          // value below, but it's never referenced.
          int32_t dest_state_idx1_old = arc->u.dest_info_state_idx1,
                 dest_state_idx01_old = next_states_idx0x_old + dest_state_idx1_old,
                 dest_state_idx01_new = -1,
                  dest_state_idx1_new = -1;
          float backward_loglike = minus_inf;
          char keep_this_arc = 0;
          if (dest_state_idx1_old == -1 ||
              next_states_old2new_data[dest_state_idx01_old + 1] ==
              next_states_old2new_data[dest_state_idx01_old]) {
            // dest_state_idx1_old == -1 means this arc was already pruned in
            // the forward pass.. do nothing.
            // If the second half of the || is true, it means the dest-state of
            // this arc was pruned away.
            // Do nothing.
          } else {
            dest_state_idx01_new = next_states_old2new_data[dest_state_idx01_old];
            dest_state_idx1_new = dest_state_idx01_new - next_states_idx0x_new;
            float arc_loglike = arc->arc_loglike;
            float dest_state_backward_loglike =
                next_states_data_new[dest_state_idx01_new].backward_loglike;
            // 'backward_loglike' is the loglike at the beginning of the arc
            backward_loglike = arc_loglike + dest_state_backward_loglike;
            float src_state_forward_loglike = OrderedIntToFloat(
                cur_states_data[arcs_rowids2[arcs_idx012]].forward_loglike);
            if (backward_loglike + src_state_forward_loglike >= -output_beam) {
              keep_this_arc = 1;
            } else {
              backward_loglike = minus_inf;  // Don't let arcs outside beam
                                             // contribute to their
                                             // start-states's backward prob
                                             // (we'll use that to prune them
                                             // away.)
            }
          }
          keep_cur_arcs_data[arcs_idx012] = keep_this_arc;
          // Correct the dest-state in the arc for the pruning of the next
          // frame's states.
          // We store the idx1 rather than the idx01 because it's more convenient
          // later on while formatting the output, although it's a little
          // inconvenient for this operation.
          arc->u.dest_info_state_idx1 = dest_state_idx1_new;
          arc_backward_prob_data[arcs_idx012] = backward_loglike;
    };
    // TODO: after debugging, maybe use K2_EVAL.
    Eval(c_, num_arcs, lambda_set_arc_backward_prob_and_keep);

    /* note, the elements of state_backward_prob that don't have arcs leaving
       them will be set to the supplied default.  */
    Array1<float> state_backward_prob(c_, num_states);
    MaxPerSublist(arc_backward_prob, minus_inf, &state_backward_prob);

    const float *state_backward_prob_data = state_backward_prob.Data();
    const int32_t *cur_states_row_ids1 =
        cur_frame->states.shape.RowIds(1).Data();

    int32_t num_fsas = NumFsas();
    K2_DCHECK_EQ(cur_frame->states.shape.Dim0(), num_fsas);
    K2_EVAL(
        c_, cur_frame->states.NumElements(), lambda_set_state_backward_prob,
        (int32_t state_idx01)->void {
          StateInfo *info = cur_states_data + state_idx01;
          int32_t fsas_state_idx01 = info->a_fsas_state_idx01,
                  a_fsas_idx0 = a_fsas_row_ids1_data[fsas_state_idx01],
                  fsas_state_idx0x_next = a_fsas_row_splits1_data[a_fsas_idx0 + 1];
          float forward_loglike = OrderedIntToFloat(info->forward_loglike),
                backward_loglike;
          // `is_final_state` means this is the final-state in a_fsas.  this
          // implies it's final in b_fsas too, since they both would have seen
          // symbols -1.
          int32_t is_final_state =
              (fsas_state_idx01 + 1 >= fsas_state_idx0x_next);
          if (is_final_state) {
            // Note: there is only one final-state.
            backward_loglike = -forward_loglike;
          } else {
            backward_loglike = state_backward_prob_data[state_idx01];
          }
          info->backward_loglike = backward_loglike;
        });

    K2_LOG(INFO) << "For t = " << t << ", renumber_cur_arcs.Keep() = "
                 << renumber_cur_arcs.Keep() << ", cur_arcs = "
                 << cur_frame->arcs;

    cur_frame->arcs = SubsampleRagged(cur_frame->arcs,
                                      renumber_cur_arcs);
  }

  ContextPtr c_;
  FsaVec &a_fsas_;         // Note: a_fsas_ has 3 axes.
  int32_t a_fsas_stride_;  // 1 if we use a different FSA per sequence
                           // (a_fsas_.Dim0() > 1), 0 if the decoding graph is
                           // shared (a_fsas_.Dim0() == 1).
  DenseFsaVec &b_fsas_;
  int32_t T_;  // == b_fsas_.MaxSize(1).
  float search_beam_;
  float output_beam_;
  int32_t min_active_;
  int32_t max_active_;
  Array1<float> dynamic_beams_;  // dynamic beams (initially just search_beam_
                                 // but change due to max_active/min_active
                                 // constraints).

  int32_t state_map_fsa_stride_;  // state_map_fsa_stride_ is a_fsas_.TotSize(1)
                                  // if a_fsas_.Dim0() == 1, else 0.

  Hash32 state_map_;  // state_map_ maps from:
                      // key == (state_map_fsa_stride_*n) + a_fsas_state_idx01,
                      //    where n is the fsa_idx, i.e. the index into b_fsas_
                      // to
                      // value, where at different stages of PropagateForward(),
                      // value is an arc_idx012 (into cur_frame->arcs), and
                      // then later a state_idx01 into the next frame's `state`
                      // member.

  // The 1st dim is needed because If all the
  // streams share the same FSA in a_fsas_, we need
  // separate maps for each).  This map is used on
  // each frame to compute and store the mapping
  // from active states to the position in the
  // `states` array.  Between frames, all values
  // have -1 in them.

  std::vector<std::unique_ptr<FrameInfo>> frames_;

};

void IntersectDensePruned(FsaVec &a_fsas, DenseFsaVec &b_fsas,
                          float search_beam, float output_beam,
                          int32_t min_active_states, int32_t max_active_states,
                          FsaVec *out, Array1<int32_t> *arc_map_a,
                          Array1<int32_t> *arc_map_b) {
  NVTX_RANGE("IntersectDensePruned");
  FsaVec a_vec = FsaToFsaVec(a_fsas);
  MultiGraphDenseIntersectPruned intersector(a_vec, b_fsas, search_beam,
                                             output_beam, min_active_states,
                                             max_active_states);

  intersector.Intersect();
  intersector.FormatOutput(out, arc_map_a, arc_map_b);
}
}  // namespace k2
